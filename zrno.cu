
#include <hip/hip_runtime.h>
#include <math.h>

__device__ float fact_fun(int idx){
  float fact = 1;
  for(int i = 1; i<idx+1; i++){
    fact = fact*i;
  }
  fact = 1/fact;
  return fact;
}

__global__ void e_sum(float *c){
  int duljina = 500;
  const int idx = threadIdx.x;
  c[idx] = fact_fun(idx);
  c[duljina-idx-1] = fact_fun(duljina-idx);

}

