                                         

#include <hip/hip_runtime.h>
#include <math.h>

__global__ void e_sum(float *c){

  const int idx = threadIdx.x;
  double fact = 1;
  for(int i = 1; i<idx+1; i++){
    fact = fact*i;
  }
  fact = 1/fact;
  c[idx] = fact;

}


